
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU!\n");
}

int main()
{
    printf("Hello World from CPU!\n");

    helloFromGPU <<<1, 10>>>();
    // cudaDeviceReset();
    hipDeviceSynchronize();
    return 0;
}
