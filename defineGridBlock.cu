#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv)
{
    int i_elem = 1024;
    dim3 block(1024);
    dim3 grid ((i_elem + block.x - 1) / block.x);
    printf("grid.x %d, block.x %d \n", grid.x, block.x);

    block.x = 512;
    grid.x = (i_elem + block.x - 1) / block.x;
    printf("grid.x %d, block.x %d \n", grid.x, block.x);

    block.x = 256;
    grid.x = (i_elem + block.x - 1) / block.x;
    printf("grid.x %d, block.x %d \n", grid.x, block.x);

    block.x = 128;
    grid.x = (i_elem + block.x - 1) / block.x;
    printf("grid.x %d, block.x %d \n", grid.x, block.x);

    hipDeviceReset();

    return 0;

}